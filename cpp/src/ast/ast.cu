#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/ast/ast.cuh>
#include <cudf/ast/linearizer.cuh>
#include <cudf/ast/operators.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <rmm/device_uvector.hpp>

#include <algorithm>
#include <functional>
#include <iterator>
#include <type_traits>

namespace cudf {

namespace ast {

namespace detail {

template <size_type block_size>
__launch_bounds__(block_size) __global__
  void compute_column_kernel(const table_device_view table,
                             const cudf::detail::fixed_width_scalar_device_view_base* literals,
                             mutable_column_device_view output_column,
                             const detail::device_data_reference* data_references,
                             const ast_operator* operators,
                             const cudf::size_type* operator_source_indices,
                             cudf::size_type num_operators,
                             cudf::size_type num_intermediates)
{
  extern __shared__ std::int64_t intermediate_storage[];
  auto thread_intermediate_storage = &intermediate_storage[threadIdx.x * num_intermediates];
  const cudf::size_type start_idx  = threadIdx.x + blockIdx.x * blockDim.x;
  const cudf::size_type stride     = blockDim.x * gridDim.x;
  auto const num_rows              = table.num_rows();
  auto const evaluator =
    cudf::ast::detail::row_evaluator(table, literals, thread_intermediate_storage, &output_column);

  for (cudf::size_type row_index = start_idx; row_index < num_rows; row_index += stride) {
    evaluate_row_expression(
      evaluator, data_references, operators, operator_source_indices, num_operators, row_index);
  }
}

template <typename T>
rmm::device_uvector<T> async_create_device_data(std::vector<T> host_data, hipStream_t stream)
{
  auto device_data = rmm::device_uvector<T>(host_data.size(), stream);
  CUDA_TRY(hipMemcpyAsync(device_data.data(),
                           host_data.data(),
                           sizeof(T) * host_data.size(),
                           hipMemcpyHostToDevice,
                           stream));
  return device_data;
}

std::unique_ptr<column> compute_column(table_view const table,
                                       expression const& expr,
                                       hipStream_t stream,
                                       rmm::mr::device_memory_resource* mr)
{
  // Linearize the AST
  auto const expr_linearizer         = linearizer(expr, table);
  auto const data_references         = expr_linearizer.get_data_references();
  auto const literals                = expr_linearizer.get_literals();
  auto const operators               = expr_linearizer.get_operators();
  auto const num_operators           = cudf::size_type(operators.size());
  auto const operator_source_indices = expr_linearizer.get_operator_source_indices();
  auto const expr_data_type          = expr_linearizer.get_root_data_type();

  // Create device data
  auto const device_data_references = detail::async_create_device_data(data_references, stream);
  auto const device_literals        = detail::async_create_device_data(literals, stream);
  auto const device_operators       = detail::async_create_device_data(operators, stream);
  auto const device_operator_source_indices =
    detail::async_create_device_data(operator_source_indices, stream);
  // The stream is synced later when the table_device_view is created.
  // To reduce overhead, we don't call a stream sync here.

  // Create table device view
  auto table_device         = table_device_view::create(table, stream);
  auto const table_num_rows = table.num_rows();

  // Prepare output column
  auto output_column = cudf::make_fixed_width_column(
    expr_data_type, table_num_rows, mask_state::UNALLOCATED, stream, mr);
  auto mutable_output_device =
    cudf::mutable_column_device_view::create(output_column->mutable_view(), stream);

  // Configure kernel parameters
  auto constexpr block_size = 512;
  cudf::detail::grid_1d config(table_num_rows, block_size);
  auto const num_intermediates = expr_linearizer.get_intermediate_count();
  auto const shmem_size_per_block =
    sizeof(std::int64_t) * num_intermediates * config.num_threads_per_block;

  // Output linearizer info
  /*
  std::cout << "LINEARIZER INFO:" << std::endl;
  std::cout << "Number of data references: " << data_references.size() << std::endl;
  std::cout << "Data references: ";
  for (auto const& dr : data_references) {
    switch (dr.reference_type) {
      case detail::device_data_reference_type::COLUMN:
        if (dr.table_reference == table_reference::LEFT) {
          std::cout << "C";
        } else {
          std::cout << "O";
        }
        break;
      case detail::device_data_reference_type::LITERAL: std::cout << "L"; break;
      case detail::device_data_reference_type::INTERMEDIATE: std::cout << "I";
    }
    std::cout << dr.data_index << "[typeid=" << static_cast<int>(dr.data_type.id()) << "], ";
  }
  std::cout << std::endl;
  std::cout << "Number of operators: " << num_operators << std::endl;
  std::cout << "Number of operator source indices: " << operator_source_indices.size() << std::endl;
  std::cout << "Number of literals: " << literals.size() << std::endl;
  std::cout << "Operator source indices: ";
  for (auto const& v : operator_source_indices) { std::cout << v << ", "; }
  std::cout << std::endl;
  std::cout << "Requesting " << config.num_blocks << " blocks, ";
  std::cout << config.num_threads_per_block << " threads/block, ";
  std::cout << shmem_size_per_block << " bytes of shared memory." << std::endl;
  */

  // Execute the kernel
  cudf::ast::detail::compute_column_kernel<block_size>
    <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream>>>(
      *table_device,
      device_literals.data(),
      *mutable_output_device,
      device_data_references.data(),
      device_operators.data(),
      device_operator_source_indices.data(),
      num_operators,
      num_intermediates);
  CHECK_CUDA(stream);
  return output_column;
}

}  // namespace detail

std::unique_ptr<column> compute_column(table_view const table,
                                       expression const& expr,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_column(table, expr, 0, mr);
}

}  // namespace ast

}  // namespace cudf
#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/ast/ast.cuh>
#include <cudf/ast/linearizer.hpp>
#include <cudf/ast/operators.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>

#include <algorithm>
#include <functional>
#include <iterator>
#include <type_traits>

namespace cudf {

namespace ast {

namespace detail {

template <size_type block_size>
__launch_bounds__(block_size) __global__
  void compute_column_kernel(table_device_view const table,
                             const cudf::detail::fixed_width_scalar_device_view_base* literals,
                             mutable_column_device_view output_column,
                             const detail::device_data_reference* data_references,
                             const ast_operator* operators,
                             const cudf::size_type* operator_source_indices,
                             cudf::size_type num_operators,
                             cudf::size_type num_intermediates)
{
  extern __shared__ std::int64_t intermediate_storage[];
  auto thread_intermediate_storage = &intermediate_storage[threadIdx.x * num_intermediates];
  auto const start_idx             = cudf::size_type(threadIdx.x + blockIdx.x * blockDim.x);
  auto const stride                = cudf::size_type(blockDim.x * gridDim.x);
  auto const num_rows              = table.num_rows();
  auto const evaluator =
    cudf::ast::detail::row_evaluator(table, literals, thread_intermediate_storage, &output_column);

  for (cudf::size_type row_index = start_idx; row_index < num_rows; row_index += stride) {
    evaluate_row_expression(
      evaluator, data_references, operators, operator_source_indices, num_operators, row_index);
  }
}

std::unique_ptr<column> compute_column(table_view const table,
                                       expression const& expr,
                                       hipStream_t stream,
                                       rmm::mr::device_memory_resource* mr)
{
  // Linearize the AST
  auto const expr_linearizer         = linearizer(expr, table);
  auto const data_references         = expr_linearizer.get_data_references();
  auto const literals                = expr_linearizer.get_literals();
  auto const operators               = expr_linearizer.get_operators();
  auto const num_operators           = cudf::size_type(operators.size());
  auto const operator_source_indices = expr_linearizer.get_operator_source_indices();
  auto const expr_data_type          = expr_linearizer.get_root_data_type();

  // Create ast_plan and device buffer
  auto plan = ast_plan();
  plan.add_to_plan(data_references);
  plan.add_to_plan(literals);
  plan.add_to_plan(operators);
  plan.add_to_plan(operator_source_indices);
  auto const host_data_buffer = plan.get_host_data_buffer();
  auto const buffer_offsets   = plan.get_offsets();
  auto const buffer_size      = host_data_buffer.second;
  auto device_data_buffer     = rmm::device_buffer(buffer_size, stream, mr);
  CUDA_TRY(hipMemcpyAsync(device_data_buffer.data(),
                           host_data_buffer.first.get(),
                           buffer_size,
                           hipMemcpyHostToDevice,
                           stream));
  // To reduce overhead, we don't call a stream sync here.
  // The stream is synced later when the table_device_view is created.

  // Create device pointers to components of plan
  auto const device_data_buffer_ptr = static_cast<const char*>(device_data_buffer.data());
  auto const device_data_references = reinterpret_cast<const detail::device_data_reference*>(
    device_data_buffer_ptr + buffer_offsets.at(0));
  auto const device_literals =
    reinterpret_cast<const cudf::detail::fixed_width_scalar_device_view_base*>(
      device_data_buffer_ptr + buffer_offsets.at(1));
  auto const device_operators =
    reinterpret_cast<const ast_operator*>(device_data_buffer_ptr + buffer_offsets.at(2));
  auto const device_operator_source_indices =
    reinterpret_cast<const cudf::size_type*>(device_data_buffer_ptr + buffer_offsets.at(3));

  // Create table device view
  auto table_device         = table_device_view::create(table, stream);
  auto const table_num_rows = table.num_rows();

  // Prepare output column
  auto output_column = cudf::make_fixed_width_column(
    expr_data_type, table_num_rows, mask_state::UNALLOCATED, stream, mr);
  auto mutable_output_device =
    cudf::mutable_column_device_view::create(output_column->mutable_view(), stream);

  // Configure kernel parameters
  auto const num_intermediates     = expr_linearizer.get_intermediate_count();
  auto const shmem_size_per_thread = static_cast<int>(sizeof(std::int64_t) * num_intermediates);
  int device_id;
  CUDA_TRY(hipGetDevice(&device_id));
  int shmem_per_block_limit;
  CUDA_TRY(
    hipDeviceGetAttribute(&shmem_per_block_limit, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id));
  auto constexpr DEFAULT_BLOCK_SIZE = 512;
  auto const block_size =
    (shmem_size_per_thread > 0)
      ? std::min(DEFAULT_BLOCK_SIZE, shmem_per_block_limit / shmem_size_per_thread)
      : DEFAULT_BLOCK_SIZE;
  cudf::detail::grid_1d config(table_num_rows, block_size);
  auto const shmem_size_per_block = shmem_size_per_thread * config.num_threads_per_block;

  // Output linearizer info
  /*
  std::cout << "LINEARIZER INFO:" << std::endl;
  std::cout << "Number of data references: " << data_references.size() << std::endl;
  std::cout << "Data references: ";
  for (auto const& dr : data_references) {
    switch (dr.reference_type) {
      case detail::device_data_reference_type::COLUMN:
        if (dr.table_source == table_reference::LEFT) {
          std::cout << "C";
        } else {
          std::cout << "O";
        }
        break;
      case detail::device_data_reference_type::LITERAL: std::cout << "L"; break;
      case detail::device_data_reference_type::INTERMEDIATE: std::cout << "I";
    }
    std::cout << dr.data_index << "[typeid=" << static_cast<int>(dr.data_type.id()) << "], ";
  }
  std::cout << std::endl;
  std::cout << "Number of operators: " << num_operators << std::endl;
  std::cout << "Number of operator source indices: " << operator_source_indices.size() << std::endl;
  std::cout << "Number of literals: " << literals.size() << std::endl;
  std::cout << "Operator source indices: ";
  for (auto const& v : operator_source_indices) { std::cout << v << ", "; }
  std::cout << std::endl;
  std::cout << "Requesting " << config.num_blocks << " blocks, ";
  std::cout << config.num_threads_per_block << " threads/block, ";
  std::cout << shmem_size_per_block << " bytes of shared memory." << std::endl;
  */

  // Execute the kernel
  cudf::ast::detail::compute_column_kernel<DEFAULT_BLOCK_SIZE>
    <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream>>>(
      *table_device,
      device_literals,
      *mutable_output_device,
      device_data_references,
      device_operators,
      device_operator_source_indices,
      num_operators,
      num_intermediates);
  CHECK_CUDA(stream);
  return output_column;
}

}  // namespace detail

std::unique_ptr<column> compute_column(table_view const table,
                                       expression const& expr,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_column(table, expr, 0, mr);
}

}  // namespace ast

}  // namespace cudf
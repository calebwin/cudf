#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/ast/ast.cuh>
#include <cudf/ast/operators.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>

#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/table_utilities.hpp>
#include <tests/utilities/type_lists.hpp>

#include <limits>
#include <type_traits>

template <typename T>
using column_wrapper = cudf::test::fixed_width_column_wrapper<T>;

struct ASTTest : public cudf::test::BaseFixture {
};

TEST_F(ASTTest, BasicAddition)
{
  auto c_0   = column_wrapper<int32_t>{3, 20, 1, 50};
  auto c_1   = column_wrapper<int32_t>{10, 7, 20, 0};
  auto table = cudf::table_view{{c_0, c_1}};

  auto col_ref_0  = cudf::ast::column_reference(0);
  auto col_ref_1  = cudf::ast::column_reference(1);
  auto expression = cudf::ast::expression(cudf::ast::ast_operator::ADD, col_ref_0, col_ref_1);

  auto expected = column_wrapper<int32_t>{13, 27, 21, 50};
  auto result   = cudf::ast::compute_column(table, expression);

  cudf::test::expect_columns_equal(expected, result->view(), true);
}

TEST_F(ASTTest, LessComparator)
{
  auto c_0   = column_wrapper<int32_t>{3, 20, 1, 50};
  auto c_1   = column_wrapper<int32_t>{10, 7, 20, 0};
  auto table = cudf::table_view{{c_0, c_1}};

  auto col_ref_0  = cudf::ast::column_reference(0);
  auto col_ref_1  = cudf::ast::column_reference(1);
  auto expression = cudf::ast::expression(cudf::ast::ast_operator::LESS, col_ref_0, col_ref_1);

  auto expected = column_wrapper<bool>{true, false, true, false};
  auto result   = cudf::ast::compute_column(table, expression);

  cudf::test::expect_columns_equal(expected, result->view(), true);
}

TEST_F(ASTTest, MultiLevelTreeArithmetic)
{
  auto c_0   = column_wrapper<int32_t>{3, 20, 1, 50};
  auto c_1   = column_wrapper<int32_t>{10, 7, 20, 0};
  auto c_2   = column_wrapper<int32_t>{-3, 66, 2, -99};
  auto table = cudf::table_view{{c_0, c_1, c_2}};

  auto col_ref_0 = cudf::ast::column_reference(0);
  auto col_ref_1 = cudf::ast::column_reference(1);
  auto col_ref_2 = cudf::ast::column_reference(2);

  auto expression_left_subtree =
    cudf::ast::expression(cudf::ast::ast_operator::ADD, col_ref_0, col_ref_1);

  auto expression_right_subtree =
    cudf::ast::expression(cudf::ast::ast_operator::SUB, col_ref_2, col_ref_0);

  auto expression_tree = cudf::ast::expression(
    cudf::ast::ast_operator::ADD, expression_left_subtree, expression_right_subtree);

  auto result   = cudf::ast::compute_column(table, expression_tree);
  auto expected = column_wrapper<int32_t>{7, 73, 22, -99};

  cudf::test::expect_columns_equal(expected, result->view(), true);
}

TEST_F(ASTTest, ImbalancedTreeArithmetic)
{
  auto c_0   = column_wrapper<double>{0.15, 0.37, 4.2, 21.3};
  auto c_1   = column_wrapper<double>{0.0, -42.0, 1.0, 98.6};
  auto c_2   = column_wrapper<double>{0.6, std::numeric_limits<double>::infinity(), 0.999, 1.0};
  auto table = cudf::table_view{{c_0, c_1, c_2}};

  auto col_ref_0 = cudf::ast::column_reference(0);
  auto col_ref_1 = cudf::ast::column_reference(1);
  auto col_ref_2 = cudf::ast::column_reference(2);

  auto expression_right_subtree =
    cudf::ast::expression(cudf::ast::ast_operator::MUL, col_ref_0, col_ref_1);

  auto expression_tree =
    cudf::ast::expression(cudf::ast::ast_operator::SUB, col_ref_2, expression_right_subtree);

  auto result = cudf::ast::compute_column(table, expression_tree);
  auto expected =
    column_wrapper<double>{0.6, std::numeric_limits<double>::infinity(), -3.201, -2099.18};

  cudf::test::expect_columns_equal(expected, result->view(), true);
}

TEST_F(ASTTest, MultiLevelTreeComparator)
{
  auto c_0   = column_wrapper<int32_t>{3, 20, 1, 50};
  auto c_1   = column_wrapper<int32_t>{10, 7, 20, 0};
  auto c_2   = column_wrapper<int32_t>{-3, 66, 2, -99};
  auto table = cudf::table_view{{c_0, c_1, c_2}};

  auto col_ref_0 = cudf::ast::column_reference(0);
  auto col_ref_1 = cudf::ast::column_reference(1);
  auto col_ref_2 = cudf::ast::column_reference(2);

  auto expression_left_subtree =
    cudf::ast::expression(cudf::ast::ast_operator::GREATER_EQUAL, col_ref_0, col_ref_1);

  auto expression_right_subtree =
    cudf::ast::expression(cudf::ast::ast_operator::GREATER, col_ref_2, col_ref_0);

  auto expression_tree = cudf::ast::expression(
    cudf::ast::ast_operator::LOGICAL_AND, expression_left_subtree, expression_right_subtree);

  auto result   = cudf::ast::compute_column(table, expression_tree);
  auto expected = column_wrapper<bool>{false, true, false, false};

  cudf::test::expect_columns_equal(expected, result->view(), true);
}

TEST_F(ASTTest, MultiTypeOperationFailure)
{
  auto c_0   = column_wrapper<int32_t>{3, 20, 1, 50};
  auto c_1   = column_wrapper<double>{0.15, 0.77, 4.2, 21.3};
  auto table = cudf::table_view{{c_0, c_1}};

  auto col_ref_0 = cudf::ast::column_reference(0);
  auto col_ref_1 = cudf::ast::column_reference(1);

  auto expression_0_plus_1 =
    cudf::ast::expression(cudf::ast::ast_operator::ADD, col_ref_0, col_ref_1);
  auto expression_1_plus_0 =
    cudf::ast::expression(cudf::ast::ast_operator::ADD, col_ref_1, col_ref_0);

  // Operations on different types are not allowed
  EXPECT_THROW(cudf::ast::compute_column(table, expression_0_plus_1), cudf::logic_error);
  EXPECT_THROW(cudf::ast::compute_column(table, expression_1_plus_0), cudf::logic_error);
}

TEST_F(ASTTest, LiteralComparison)
{
  auto c_0   = column_wrapper<int32_t>{3, 20, 1, 50};
  auto table = cudf::table_view{{c_0}};

  auto col_ref_0     = cudf::ast::column_reference(0);
  auto literal_value = cudf::numeric_scalar<int32_t>(41);
  auto literal_view  = cudf::get_scalar_device_view(literal_value);
  auto literal       = cudf::ast::literal(literal_view);

  auto expression = cudf::ast::expression(cudf::ast::ast_operator::GREATER, col_ref_0, literal);

  auto result   = cudf::ast::compute_column(table, expression);
  auto expected = column_wrapper<bool>{false, false, false, true};

  cudf::test::expect_columns_equal(expected, result->view(), true);
}

TEST_F(ASTTest, UnaryNot)
{
  auto c_0   = column_wrapper<int32_t>{3, 0, 1, 50};
  auto table = cudf::table_view{{c_0}};

  auto col_ref_0 = cudf::ast::column_reference(0);

  auto expression = cudf::ast::expression(cudf::ast::ast_operator::NOT, col_ref_0);

  auto result   = cudf::ast::compute_column(table, expression);
  auto expected = column_wrapper<bool>{false, true, false, false};

  cudf::test::expect_columns_equal(expected, result->view(), true);
}

TEST_F(ASTTest, UnaryTrigonometry)
{
  auto c_0   = column_wrapper<double>{0.0, M_PI / 4, M_PI / 3};
  auto table = cudf::table_view{{c_0}};

  auto col_ref_0 = cudf::ast::column_reference(0);

  auto expected_sin   = column_wrapper<double>{0.0, std::sqrt(2) / 2, std::sqrt(3.0) / 2.0};
  auto expression_sin = cudf::ast::expression(cudf::ast::ast_operator::SIN, col_ref_0);
  auto result_sin     = cudf::ast::compute_column(table, expression_sin);
  cudf::test::expect_columns_equivalent(expected_sin, result_sin->view(), true);

  auto expected_cos   = column_wrapper<double>{1.0, std::sqrt(2) / 2, 0.5};
  auto expression_cos = cudf::ast::expression(cudf::ast::ast_operator::COS, col_ref_0);
  auto result_cos     = cudf::ast::compute_column(table, expression_cos);
  cudf::test::expect_columns_equivalent(expected_cos, result_cos->view(), true);

  auto expected_tan   = column_wrapper<double>{0.0, 1.0, std::sqrt(3.0)};
  auto expression_tan = cudf::ast::expression(cudf::ast::ast_operator::TAN, col_ref_0);
  auto result_tan     = cudf::ast::compute_column(table, expression_tan);
  cudf::test::expect_columns_equivalent(expected_tan, result_tan->view(), true);
}

TEST_F(ASTTest, ArityCheckFailure)
{
  auto col_ref_0 = cudf::ast::column_reference(0);
  EXPECT_THROW(cudf::ast::expression(cudf::ast::ast_operator::ADD, col_ref_0), cudf::logic_error);
  EXPECT_THROW(cudf::ast::expression(cudf::ast::ast_operator::ABS, col_ref_0, col_ref_0),
               cudf::logic_error);
}

TEST_F(ASTTest, StringComparison)
{
  auto c_0   = cudf::test::strings_column_wrapper({"a", "bb", "ccc", "dddd"});
  auto c_1   = cudf::test::strings_column_wrapper({"aa", "b", "cccc", "ddd"});
  auto table = cudf::table_view{{c_0, c_1}};

  auto col_ref_0  = cudf::ast::column_reference(0);
  auto col_ref_1  = cudf::ast::column_reference(1);
  auto expression = cudf::ast::expression(cudf::ast::ast_operator::LESS, col_ref_0, col_ref_1);

  auto expected = column_wrapper<bool>{true, false, true, false};
  auto result   = cudf::ast::compute_column(table, expression);

  cudf::test::expect_columns_equal(expected, result->view(), true);
}

TEST_F(ASTTest, CopyColumn)
{
  auto c_0   = column_wrapper<int32_t>{3, 0, 1, 50};
  auto table = cudf::table_view{{c_0}};

  auto col_ref_0  = cudf::ast::column_reference(0);
  auto expression = cudf::ast::expression(cudf::ast::ast_operator::IDENTITY, col_ref_0);

  auto result   = cudf::ast::compute_column(table, expression);
  auto expected = column_wrapper<int32_t>{3, 0, 1, 50};

  cudf::test::expect_columns_equal(expected, result->view(), true);
}

TEST_F(ASTTest, CopyLiteral)
{
  auto c_0   = column_wrapper<int32_t>{0, 0, 0, 0};
  auto table = cudf::table_view{{c_0}};

  auto literal_value = cudf::numeric_scalar<int32_t>(-123);
  auto literal_view  = cudf::get_scalar_device_view(literal_value);
  auto literal       = cudf::ast::literal(literal_view);

  auto expression = cudf::ast::expression(cudf::ast::ast_operator::IDENTITY, literal);

  auto result   = cudf::ast::compute_column(table, expression);
  auto expected = column_wrapper<int32_t>{-123, -123, -123, -123};

  cudf::test::expect_columns_equal(expected, result->view(), true);
}

struct custom_functor {
  template <typename OperatorFunctor,
            typename LHS,
            typename RHS,
            std::enable_if_t<cudf::ast::is_valid_binary_op<OperatorFunctor, LHS, RHS>>* = nullptr>
  CUDA_HOST_DEVICE_CALLABLE decltype(auto) operator()(int* result)
  {
    *result = 42;
  }

  template <typename OperatorFunctor,
            typename LHS,
            typename RHS,
            std::enable_if_t<!cudf::ast::is_valid_binary_op<OperatorFunctor, LHS, RHS>>* = nullptr>
  CUDA_HOST_DEVICE_CALLABLE decltype(auto) operator()(int* result)
  {
  }
};

TEST_F(ASTTest, CustomASTFunctor)
{
  int result = 0;
  cudf::ast::binary_operator_dispatcher(cudf::ast::ast_operator::ADD,
                                        cudf::data_type(cudf::type_id::INT32),
                                        cudf::data_type(cudf::type_id::INT32),
                                        custom_functor{},
                                        &result);
  EXPECT_EQ(result, 42);
}

CUDF_TEST_PROGRAM_MAIN()
